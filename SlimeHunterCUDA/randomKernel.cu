#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include "jrandom.cuh"

__global__ void genRandomKernel(int64_t *out, const int64_t *in, Random *random, const size_t n) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n) {
		return;
	}
	out[i] = nextLongKernel(setSeedKernel(&random[i], in[i]));
	//out[i] = in[i];
}

hipError_t randomWithCuda(int64_t * out, int64_t *in, Random* random, size_t n) {
	int64_t *dev_in = NULL;
	int64_t *dev_out = NULL;
	Random *dev_random = NULL;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed\n");
		goto error;
	}

	cudaStatus = hipMalloc(&dev_out, n * sizeof(int64_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed(dev_out)\n");
		goto error;
	}

	cudaStatus = hipMalloc(&dev_in, n * sizeof(int64_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed(dev_in)\n");
		goto error;
	}

	cudaStatus = hipMalloc(&dev_random, n * sizeof(Random));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed(dev_in)\n");
		goto error;
	}

	cudaStatus = hipMemcpy(dev_in, in, n * sizeof(int64_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed(dev_in)\n");
		goto error;
	}

	cudaStatus = hipMemcpy(dev_random, random, n * sizeof(Random), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed(dev_in)\n");
		goto error;
	}

	genRandomKernel << <1, n >> > (dev_out, dev_in, dev_random, n);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(out, dev_out, n * sizeof(int64_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto error;
	}
error:
	hipFree(dev_in);
	hipFree(dev_out);
	hipFree(dev_random);
	return cudaStatus;
}
