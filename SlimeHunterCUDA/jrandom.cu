
#include <hip/hip_runtime.h>
#include <>
#include "jrandom.cuh"

__device__ static int64_t initialScrambleKernel(uint64_t seed) {
	return (seed ^ MULTIPLIER) & MASK;
}

__device__ Random* setSeedKernel(Random *rnd, int64_t seed) {
	rnd->seed = initialScrambleKernel(seed);
	return rnd;
}

__device__ int32_t nextKernel(Random* rnd, int32_t bits) {
	uint64_t oldseed, nextseed;
	uint64_t seed = rnd->seed;
	oldseed = seed;
	nextseed = (oldseed * MULTIPLIER + ADDEND) & MASK;
	rnd->seed = nextseed;
	return (uint32_t)(nextseed >> (48 - bits));
}

__device__ int64_t nextLongKernel(Random* rnd) {
	return ((int64_t)(nextKernel(rnd, 32)) << 32) + nextKernel(rnd, 32);
}

__device__ int32_t nextIntKernel(Random* rnd) {
	return nextKernel(rnd, 32);
}

__device__ int32_t nextIntWithRangeKernel(Random *rnd, int32_t bound) {
	if (bound <= 0) {
		// err!
		return 0;
	}
	int32_t r = nextKernel(rnd, 31);
	int32_t m = bound - 1;
	if ((bound & m) == 0) {
		r = (int32_t)((bound * (int64_t)r) >> 31);
	}
	else {
		int32_t u;
		for (u = r; u - (r = u % bound) + m < 0; u = nextKernel(rnd, 31));
	}
	return r;
}
