
#include <hip/hip_runtime.h>
#include <>
#include "mcSlimeChunkOracle.cuh"

#if 1
__device__ SlimeChunkSeed* setMCSeedKernel(SlimeChunkSeed* t, int64_t seed) {
	t->seed = seed;
	return t;
}

__device__ int64_t getMCSeedKernel(SlimeChunkSeed* seed, int32_t chunkX, int32_t chunkZ) {
	return seed->seed + chunkX * chunkX * 0x4c1906 + chunkX * 0x5ac0db + chunkZ * chunkZ * 0x4307a7L + chunkZ * 0x5f24f ^ 0x3ad8025f;
}

__device__ static bool isSlimeChunkKernel(Random* rnd) {
	return nextIntWithRangeKernel(rnd, 10) == 0;
}

__device__ bool isSlimeChunkXZKernel(SlimeChunkSeed* seed, int64_t chunkX, int64_t chunkZ) {
	return isSlimeChunkKernel(setSeedKernel(&seed->rnd, getMCSeedKernel(seed, (int32_t)chunkX, (int32_t)chunkZ)));
}
#else

__device__ SlimeChunkSeed* setMCSeedKernel(SlimeChunkSeed* t, int64_t seed) {
	t->seed = seed;
	return t;
}

__device__ int64_t getMCSeedKernel(SlimeChunkSeed* seed, int32_t chunkX, int32_t chunkZ) {
	return seed->seed + chunkX * chunkX * 0x4c1906 + chunkX * 0x5ac0db + chunkZ * chunkZ * 0x4307a7L + chunkZ * 0x5f24f ^ 0x3ad8025f;
}

__device__ bool isSlimeChunkKernel(Random* rnd) {
	//return nextIntWithRangeKernel(rnd, 10) == 0;
	return false;
}

__device__ bool isSlimeChunkXZKernel(SlimeChunkSeed* mCSeed, int64_t chunkX, int64_t chunkZ) {
	//int64_t seed = getMCSeedKernel(mCSeed, (int32_t)chunkX, (int32_t)chunkZ);
	//Random *rnd = setSeedKernel(&mCSeed->rnd, seed);
	//return isSlimeChunkKernel(rnd);
	isSlimeChunkKernel(&mCSeed->rnd);
	return 0;
}
#endif
