
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main(int argc, char* argv[])
{
	int arraySize = 50;
	const int a[50] = {
		1, 2, 3, 4, 5 , 6, 7, 8, 9, 10,
		11, 12, 13, 14, 15, 16,17,18,19,20,
		21, 22, 23, 24, 25, 26,27,28,29,30,
		31, 32, 33, 34, 35, 36,37,38,39,40,
		41, 42, 43, 44, 45, 46,47,48,49,50
	};
	const int b[50] = {
		10, 20, 30, 40, 50 , 60, 70, 80, 90, 100,
		110, 120, 130, 140, 150, 160,170,180,190,200,
		210, 220, 230, 240, 250, 260,270,280,290,300,
		310, 320, 330, 340, 350, 360,370,380,390,400,
		410, 420, 430, 440, 450, 460,470,480,490,500
	};
	int c[50] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	printf("{1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28, 29, 30, 31, 32, 33, 34, 35, 36, 37, 38, 39, 40, 41, 42, 43, 44, 45, 46, 47, 48, 49, 50} +{ 10, 20, 30, 40, 50, 60, 70, 80, 90, 100,110, 120, 130, 140, 150, 160, 170, 180, 190, 200,210, 220, 230, 240, 250, 260, 270, 280, 290, 300,310, 320, 330, 340, 350, 360, 370, 380, 390, 400,410, 420, 430, 440, 450, 460, 470, 480, 490, 500} = {%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4], c[5], c[6], c[7], c[8], c[9],
		c[10], c[11], c[12], c[13], c[14], c[15], c[16], c[17], c[18], c[19],
		c[20], c[21], c[22], c[23], c[24], c[25], c[26], c[27], c[28], c[29],
		c[30], c[31], c[32], c[33], c[34], c[35], c[36], c[37], c[38], c[39],
		c[40], c[41], c[42], c[43], c[44], c[45], c[46], c[47], c[48], c[49]);

	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
