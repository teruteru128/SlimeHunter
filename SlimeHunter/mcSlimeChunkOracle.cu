
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "mcSlimeChunkOracle.cuh"

void setMCSeed(SlimeChunkSeed* t, int64_t seed) {
	t->seed = seed;
}

int64_t getMCSeed(SlimeChunkSeed* seed, int32_t chunkX, int32_t chunkZ) {
	return seed->seed + chunkX * chunkX * 0x4c1906 + chunkX * 0x5ac0db + chunkZ * chunkZ * 0x4307a7L + chunkZ * 0x5f24f ^ 0x3ad8025f;
}

bool isSlimeChunk(Random* rnd) {
	return nextIntWithRange(rnd, 10) == 0;
}

bool isSlimeChunkXZ(SlimeChunkSeed* seed, int64_t chunkX, int64_t chunkZ) {
	Random* rnd = &seed->rnd;
	setSeed(rnd, getMCSeed(seed, (int32_t)chunkX, (int32_t)chunkZ));
	return isSlimeChunk(rnd);
}
/**/
__device__ void setMCSeedKernel(SlimeChunkSeed* t, int64_t seed) {
	t->seed = seed;
}

__device__ int64_t getMCSeedKernel(SlimeChunkSeed* seed, int32_t chunkX, int32_t chunkZ) {
	return seed->seed + chunkX * chunkX * 0x4c1906 + chunkX * 0x5ac0db + chunkZ * chunkZ * 0x4307a7L + chunkZ * 0x5f24f ^ 0x3ad8025f;
}

__device__ bool isSlimeChunkKernel(Random* rnd) {
	return nextIntWithRangeKernel(rnd, 10) == 0;
}

__device__ bool isSlimeChunkXZKernel(SlimeChunkSeed* seed, int64_t chunkX, int64_t chunkZ) {
	Random* rnd = &seed->rnd;
	setSeedKernel(rnd, getMCSeedKernel(seed, (int32_t)chunkX, (int32_t)chunkZ));
	return isSlimeChunkKernel(rnd);
}

