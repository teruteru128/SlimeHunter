
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "mcSlimeChunkOracle.cuh"

__device__ SlimeChunkSeed* setMCSeedKernel(SlimeChunkSeed* t, int64_t seed) {
	t->seed = seed;
	return t;
}

__device__ int64_t getMCSeedKernel(SlimeChunkSeed* seed, int32_t chunkX, int32_t chunkZ) {
	return seed->seed + chunkX * chunkX * 0x4c1906 + chunkX * 0x5ac0db + chunkZ * chunkZ * 0x4307a7L + chunkZ * 0x5f24f ^ 0x3ad8025f;
}

__device__ static bool isSlimeChunkKernel(Random* rnd) {
	return nextIntWithRangeKernel(rnd, 10) == 0;
}

__device__ bool isSlimeChunkXZKernel(SlimeChunkSeed* seed, int64_t chunkX, int64_t chunkZ) {
	return isSlimeChunkKernel(setSeedKernel(&seed->rnd, getMCSeedKernel(seed, (int32_t)chunkX, (int32_t)chunkZ)));
}
